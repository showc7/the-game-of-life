/*
 *
 * compiling:
 * nvcc -lglut -LGLEW life.cuda.cu -o life
 * 
 * for it's work:
 * export LD_LIBRARY_PATH=:/usr/local/cuda/lib
 * export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/usr/local/cuda/libnvvp/
 * 
 * cuda-gdb
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define uchar unsigned char
#define NUMBER_OF_THREADS	512

uchar * dev_array1;
uchar * dev_array2;

uint * dev_size_x;
uint * dev_size_y;

uint sizeX,sizeY;

// bad -> fix it

__global__ void kernel2(float * field1, float * field2, uchar4 * screen, int sizex, int sizey)
{
//	int id = threadIdx.x + blockIdx.x*blockDim.x + threadIdx.y + blockIdx.y*blockDim.y;
	int id = threadIdx.x+threadIdx.y;
	
	int numberofneighbours=0;

	numberofneighbours += *(field1 + id + 1);
	numberofneighbours += *(field1 + id - 1);
	numberofneighbours += *(field1 + id + sizey);
	numberofneighbours += *(field1 + id - sizey);
	numberofneighbours += *(field1 + id + sizey + 1);
	numberofneighbours += *(field1 + id + sizey - 1);
	numberofneighbours += *(field1 + id - sizey - 1);
	numberofneighbours += *(field1 + id + sizey + 1);
	
//	(screen + id)->x = numberofneighbours;
//	(screen + id)->x = 1;
//	*(field2 + id) = 1;
//	if(*(field1 + id) == 1) *(field2 + id) = 1;
	/*
	switch(numberofneighbours)
	{
		case 3 :	*(field2 + id) = 1;
					(screen + id)->x = 0;
					(screen + id)->y = 250;
					(screen + id)->z = 0;
					(screen + id)->w = 0;
				break;
				
		case 2 :	if(*(field1 + id) == 1)
					{
						*(field2 + id) = 1;
						(screen + id)->x = 0;
						(screen + id)->y = 250;
						(screen + id)->z = 0;
						(screen + id)->w = 0;
					}
					else
					{
						*(field2 + id) = 0;
						(screen + id)->x = 0;
						(screen + id)->y = 0;
						(screen + id)->z = 0;
						(screen + id)->w = 0;
					}
				break;
				
		default :	*(field2 + id) = 0;
					(screen + id)->x = 0;
					(screen + id)->y = 0;
					(screen + id)->z = 0;
					(screen + id)->w = 0;
				break;
	}
	*/
}

__global__ void kernel(uchar * array1, uchar * array2, uint size_x, uint size_y)
{
//	int id = threadIdx.x*blockIdx.x+blockDim.x + threadIdx.y*blockIdx.y+blockDim.y;
	int id = threadIdx.x+threadIdx.y;
	
	if(id < size_x*size_y)
	{
		int numberofneighbours=0;
		
		int num = 0;
		
		// заменить на num += ...
		
		if(*(array1 + id + 1) == 1) num++;
		if(*(array1 + id - 1) == 1) num++;
		if(*(array1 + id + size_y) == 1) num++;
		if(*(array1 + id - size_y) == 1) num++;
		if(*(array1 + id + size_y + 1) == 1) num++;
		if(*(array1 + id + size_y - 1) == 1) num++;
		if(*(array1 + id - size_y + 1) == 1) num++;
		if(*(array1 + id - size_y - 1) == 1) num++;
		
		switch(num)
		{
			case 3 : *(array2 + id) = 1; break;
			case 2 : if(*(array1 + id) == 1) *(array2 + id) = 1; break;
			default : *(array2 + id) = 0; break;
		}
		
	}
}

void initCuda(uchar * array1, uchar * array2, uint size_x, uint size_y)
{
	sizeX = size_x;
	sizeY = size_y;
	
	hipMalloc((void**)&dev_array1,sizeof(uchar)*size_x*size_y);
	hipMalloc((void**)&dev_array2,sizeof(uchar)*size_x*size_y);
	
	hipMalloc((void**)&dev_size_x,sizeof(uint));
	hipMalloc((void**)&dev_size_y,sizeof(uint));
	
	hipMemcpy(dev_array1,array1,sizeof(uchar)*size_x*size_y,hipMemcpyHostToDevice);
	
	hipMemcpy(dev_size_x,&size_x,sizeof(uchar),hipMemcpyHostToDevice);
	hipMemcpy(dev_size_y,&size_y,sizeof(uchar),hipMemcpyHostToDevice);
	
	hipMemset(dev_array2,0,size_x*size_y);
}

void get_result(uchar * array)
{
	hipMemcpy(array,dev_array2,sizeX*sizeY*sizeof(uchar),hipMemcpyDeviceToHost);
}

void cuda_run()
{
	int threads = NUMBER_OF_THREADS;
	int blocks = sizeX*sizeY/threads+1;
	
//	kernel <<<blocks,threads>>>(dev_array1,dev_array2,dev_size_x,dev_size_y);
	kernel <<<blocks,threads>>>(dev_array1,dev_array2,sizeX,sizeY);
}

void FreeCuda()
{
	hipFree(dev_array1);
	hipFree(dev_array2);
	hipFree(dev_size_x);
	hipFree(dev_size_y);
}

void check(uchar * array1, uchar * array2, uint * size_x, uint * size_y)
{
	int num=0;
	puts("<=================>\n");
	
	printf("%u %u\n",*size_x,*size_y);
	
	puts("<=================>\n");
	
	for(int i=0;i<(*size_x * (*size_y));i++)
	{
		num = 0;
		
		if(*(array1 + i + 1) == 1) num++;
		if(*(array1 + i - 1) == 1) num++;
		if(*(array1 + i + *size_y) == 1) num++;
		if(*(array1 + i - *size_y) == 1) num++;
		if(*(array1 + i + *size_y + 1) == 1) num++;
		if(*(array1 + i + *size_y - 1) == 1) num++;
		if(*(array1 + i - *size_y + 1) == 1) num++;
		if(*(array1 + i - *size_y - 1) == 1) num++;
		
	//	*(array2 + i) = num;
		
		switch(num)
		{
			case 3 : *(array2 + i) = 1; break;
			case 2 : if(*(array1 + i) == 1) *(array2 + i) = 1; break;
			default : *(array2 + i) = 0; break;
		}
		
	}
	
	puts("<=================>\n");
}

#define cell_size 5
#define uchar unsigned char
#define screen_width 20
#define screen_height 20

int width = screen_width*cell_size; //770; //1024;
int height = screen_width*cell_size; //770; //768;
/*
int main()
{
//	uchar field1[screen_width][screen_height];
//	uchar field2[screen_width][screen_height];

	uchar field1[screen_width*screen_height];
	uchar field2[screen_width*screen_height];

	for(int i=0;i<screen_width;i++)
	{
		for(int j=0;j<screen_height;j++)
		{
			field1[i+j*screen_width] = 0;
			field2[i+j*screen_width] = 0;
		}
	}

	field1[0+0*screen_width] = 1;
	field1[0+1*screen_width] = 1;
	field1[0+2*screen_width] = 1;
//	field1[1][0] = 1;
	field1[1+1*screen_width] = 1;
	field1[3+3*screen_width] = 1;
	field1[4+3*screen_width] = 1;
	field1[10+5*screen_width] = 1;
	field1[10+6*screen_width] = 1;
	field1[11+5*screen_width] = 1;
	field1[11+6*screen_width] = 1;
//	field1[10][7] = 1;
	
	for(int i=0;i<screen_width;i++)
	{
		for(int j=0;j<screen_height;j++)
		{
			if(field1[i+j*screen_width] == 1) printf("*");
			else printf(".");
		//	printf("%c ",field1[i][j]);
		}
		printf("\n");
	}

	sizeX = 20;
	sizeY = 20;

//=======================================================================
/*
	check(field1,field2,&sizeX,&sizeY);

	for(int i=0;i<screen_width;i++)
	{
		for(int j=0;j<screen_height;j++)
		{
			if(field2[i+j*screen_width] == 1) printf("*");
			else printf("%i",(int) field2[i+j*screen_width]);
		//	printf("%c ",field2[i][j]);
		}
		printf("\n");
	}
*/
//=======================================================================
/*
	initCuda(&field1[0],&field2[0],screen_width,screen_height);
	cuda_run();
	get_result(&field2[0]);

	puts("\n<==========>\n\n");

	for(int i=0;i<screen_width;i++)
	{
		for(int j=0;j<screen_height;j++)
		{
			if(field2[i+j*screen_width] == 1) printf("*");
			else printf("%i",(int) field2[i+j*screen_width]);
		//	printf("%c ",field2[i][j]);
		}
		printf("\n");
	}

	FreeCuda();

	return 0;
}
*/
int main()
{
	float * field1;
	field1 = (float *) malloc(sizeof(float)*screen_width*screen_height);
	uchar4 * screen;
	screen = (uchar4 *) malloc(sizeof(uchar4)*screen_width*screen_height);
	
	float * dev_field1;
	float * dev_field2;
	uchar4 * dev_screen;
	
	*(field1 + 10) = 1;
	field1[40] = 1;
	field1[41] = 1;
	field1[59] = 1;
	field1[60] = 1;
	
	field1[90] = 1;
	field1[91] = 1;
	field1[92] = 1;
	
	for(int i=0;i<=400;i++)
	{
		printf("%i",(int) *(field1+i));
		if(i % 19 == 0) printf("\n");
	}
	printf("\n");
	hipMalloc((void **)&dev_field1,sizeof(float)*screen_width*screen_height);
	hipMalloc((void **)&dev_field2,sizeof(float)*screen_width*screen_height);
	hipMalloc((void **)&dev_screen,sizeof(uchar4)*screen_width*screen_height);
	
	hipMemcpy(dev_field1,field1,sizeof(uchar4)*screen_width*screen_height,hipMemcpyHostToDevice);
	
	hipMemset(dev_field2,8,sizeof(float)*screen_width*screen_height);
	
	kernel2<<<10 , 10>>>(dev_field1,dev_field2,dev_screen,20,20);
	
	float * field2;
	field2 = (float *) malloc(sizeof(float)*screen_width*screen_height);
	
	hipMemcpy(field2,dev_field2,sizeof(float)*screen_width*screen_height,hipMemcpyDeviceToHost);
	
	hipMemcpy(screen,dev_screen,sizeof(uchar4)*screen_width*screen_height,hipMemcpyDeviceToHost);
	
	hipFree(dev_field1);
	hipFree(dev_field2);
	hipFree(dev_screen);
	
	for(int i=0;i<400;i++)
	{
		printf("%i",(int) (screen+i)->x);
		if(i % 19 == 0) printf("\n");
	}
	printf("\n");
	
	for(int i=0;i<400;i++)
	{
		printf("%4i",(int) *(field2+i));
		if(i % 19 == 0) printf("\n");
	}
	printf("\n");
	
	free(field1);
	free(field2);
	free(screen);
}
