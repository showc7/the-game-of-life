/*
 *
 * compiling:
 * nvcc -lglut -LGLEW life.cuda.cu -o life -g -G
 * 
 * -g -G  - debug options
 * 
 * for it's work:
 * export LD_LIBRARY_PATH=:/usr/local/cuda/lib
 * export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/usr/local/cuda/libnvvp/
 * 
 * cuda-gdb
 */

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <memory.h>

#define FIELD_WIDTH			10
#define FIELD_HEIGHT		10
#define NUMBER_OF_THREADS	10

float * state_first;	// on PC
float * state_second;	// arrays

float * dev_first_state;	// on Card
float * dev_second_state;	// arrays

int * dev_width;
int * dev_height;

int width = FIELD_WIDTH;
int height = FIELD_HEIGHT;

__global__ void kernel(float * first, float * second , int * width, int * height)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	*(second + id) = 0;

	if(id <= (*width)*(*height))
	{
		int num = 0;
		
		// change to num += ...
		
		/*
		if(*(first + id + 1) == 1) num++;
		if(*(first + id - 1) == 1) num++;
		if(*(first + id + *height) == 1) num++;
		if(*(first + id - *height) == 1) num++;
		if(*(first + id + *height + 1) == 1) num++;
		if(*(first + id + *height - 1) == 1) num++;
		if(*(first + id - *height + 1) == 1) num++;
		if(*(first + id - *height - 1) == 1) num++;
		*/
		
		num += *(first + id + 1);
		num += *(first + id - 1);
		num += *(first + id + *height);
		num += *(first + id - *height);
		num += *(first + id + *height + 1);
		num += *(first + id + *height - 1);
		num += *(first + id - *height + 1);
		num += *(first + id - *height - 1);
		
		switch(num)
		{
			case 3 : *(second + id) = 1; break;
			case 2 : if(*(first + id) == 1) *(second + id) = 1; break;
			default : *(second + id) = 0; break;
		}
		
	}
}

void GetDataFromCudaDevice(int width, int height)
{
	hipMemcpy(state_first,dev_second_state,sizeof(float)*width*height,hipMemcpyDeviceToHost);
}

void CopyDataToCudaDevice(int width, int height)
{
	hipMemcpy(dev_first_state,state_first,sizeof(float)*width*height,hipMemcpyHostToDevice);
	hipMemset(dev_second_state,0,sizeof(float)*width*height);
	
	hipMemcpy(dev_width,&width,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_height,&height,sizeof(int),hipMemcpyHostToDevice);
}

void InitCudaArrays(int width, int height)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	hipMalloc((void**)&dev_width,sizeof(int));
	hipMalloc((void**)&dev_height,sizeof(int));

	hipMalloc((void**)&dev_first_state,sizeof(float)*width*height);
	hipMalloc((void**)&dev_second_state,sizeof(float)*width*height);
}
// runs cuda device and returns result
void RunCudaDevice()
{
	hipError_t cudaStatus;

	int threads = NUMBER_OF_THREADS;
	int blocks = (width*height)/(NUMBER_OF_THREADS + 1);
	
	kernel <<<threads,blocks>>> (dev_first_state,dev_second_state,dev_width,dev_height);
//	kernel <<<10,10>>> (dev_first_state,dev_second_state,dev_width,dev_height);

	hipDeviceSynchronize();

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	GetDataFromCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
}

void FreeCudaDevice(int width, int height)
{
	hipFree(dev_first_state);
	hipFree(dev_second_state);
	
	hipFree(dev_width);
	hipFree(dev_height);
}

void FillField()
{
	/*
	 *	
	 *	01010
	 *	00110
	 *	00100
	 *	00000
	 *	
	 */
	
//	state_first[9*width+9] = 1;

	state_first[2*width+5] = 1;
	state_first[2*width+6] = 1;
	state_first[3*width+6] = 1;
	state_first[3*width+7] = 1;
	state_first[1*width+7] = 1;

/*
	state_first[7*width+1] = 1;
	state_first[7*width+2] = 1;
	state_first[8*width+2] = 1;
	state_first[8*width+3] = 1;
	state_first[6*width+3] = 1;
*/
/*
	state_first[7*width+7] = 1;
	state_first[7*width+8] = 1;
	state_first[8*width+7] = 1;
	state_first[8*width+8] = 1;
*/
}
// allocate memory and initialize array with '0'
void InitArrays(int width, int height)
{
	state_first = (float *) malloc(sizeof(float)*width*height);
	state_second = (float *) malloc(sizeof(float)*width*height);
	
	memset(state_first,0,sizeof(float)*width*height);
	memset(state_second,0,sizeof(float)*width*height);
}

void ShowArray(int width, int height)
{
	puts("-----------------");
	for(int i=0;i<width;i++)
	{
		for(int j=0;j<height;j++)
		{
			if(state_first[i*width+j] != 0)printf("*");
			else printf(" ");
		//	printf("%1.0f",state_first[i*width+j]);
		}
		printf("\n");
	}
	puts("-----------------");
}

void CudaSwapArrays()
{
	float * t = dev_first_state;
	dev_first_state = dev_second_state;
	dev_second_state = t;
	
//	cudaMemset(dev_second_state,0,sizeof(float)*width*height); //checking
}

int main()
{
	InitArrays(FIELD_WIDTH,FIELD_HEIGHT);
	
	FillField();
	
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	
	InitCudaArrays(FIELD_WIDTH,FIELD_HEIGHT);
	CopyDataToCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
	
	RunCudaDevice();
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	
	for(int i=0;i<30;i++)
	{
		CudaSwapArrays();
		RunCudaDevice();
		ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	}
	
	FreeCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
	
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);

//	char ch;
//	scanf("%c",&ch);
}
