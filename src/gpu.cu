#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include "gpu.h"

__global__ void transfer_kernel(float * first, float * second , int * width, int * height)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	*(second + id) = 0;

	if(id <= (*width)*(*height))
	{
		int num = 0;
		
		/*
		if(*(first + id + 1) == 1) num++;
		if(*(first + id - 1) == 1) num++;
		if(*(first + id + *height) == 1) num++;
		if(*(first + id - *height) == 1) num++;
		if(*(first + id + *height + 1) == 1) num++;
		if(*(first + id + *height - 1) == 1) num++;
		if(*(first + id - *height + 1) == 1) num++;
		if(*(first + id - *height - 1) == 1) num++;
		*/
		
		num += *(first + id + 1);
		num += *(first + id - 1);
		num += *(first + id + *height);
		num += *(first + id - *height);
		num += *(first + id + *height + 1);
		num += *(first + id + *height - 1);
		num += *(first + id - *height + 1);
		num += *(first + id - *height - 1);
		
		switch(num)
		{
			case 3 : *(second + id) = 1; break;
			case 2 : if(*(first + id) == 1) *(second + id) = 1; break;
			default : *(second + id) = 0; break;
		}
	}
}

void GetDataFromCudaDevice(float * state_first, float * dev_second_state, int width, int height)
{
	#ifdef DEBUG
		puts("GetDataFromCudaDevice");
	#endif
	hipMemcpy(state_first,dev_second_state,sizeof(float)*width*height,hipMemcpyDeviceToHost);
}

void CopyDataToCudaDevice(float * dev_first_state, float * dev_second_state, float * state_first, int * dev_width, int * dev_height, int width, int height)
{
	#ifdef DEBUG
		puts("CopyDataToCudaDevice");
	#endif
	hipMemcpy(dev_first_state,state_first,sizeof(float)*width*height,hipMemcpyHostToDevice);
	hipMemset(dev_second_state,0,sizeof(float)*width*height);
	
	hipMemcpy(dev_width,&width,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_height,&height,sizeof(int),hipMemcpyHostToDevice);
}

int InitCudaArrays(float ** dev_first_state, float ** dev_second_state, int ** dev_width, int ** dev_height, int width, int height)
{
	#ifdef DEBUG
		puts("InitCudaArrays");
	#endif
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return 1;
	}

	hipMalloc((void**)&*dev_width,sizeof(int));
	hipMalloc((void**)&*dev_height,sizeof(int));

	hipMalloc((void**)&*dev_first_state,sizeof(float)*width*height);
	hipMalloc((void**)&*dev_second_state,sizeof(float)*width*height);
	
	return 0;
}

void RunCudaDevice(int threads, int blocks, float * dev_first_state, float * dev_second_state, int * dev_width, int * dev_height, int width, int height)
{
	#ifdef DEBUG
		puts("RunCudaDevice");
	#endif
	hipError_t cudaStatus;

//	int threads = NUMBER_OF_THREADS;
//	int blocks = (width*height)/(NUMBER_OF_THREADS + 1);
	
	transfer_kernel <<<threads,blocks>>> (dev_first_state,dev_second_state,dev_width,dev_height);

	hipDeviceSynchronize();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

//	GetDataFromCudaDevice(width,height);
}
void FreeCudaDevice(float * dev_first_state, float * dev_second_state, int * dev_width, int * dev_height, int width, int height)
{
	hipFree(dev_first_state);
	hipFree(dev_second_state);
	
	hipFree(dev_width);
	hipFree(dev_height);
}

void CudaSwapArrays(float ** dev_first_state, float ** dev_second_state)
{
	#ifdef DEBUG
		puts("CudaSwapArrays");
	#endif
	float * t = *dev_first_state;
	*dev_first_state = *dev_second_state;
	*dev_second_state = t;
}

