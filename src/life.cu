#include "hip/hip_runtime.h"
/*
 *
 * compiling:
 * nvcc -lglut -LGLEW life.cuda.cu -o life -g -G
 * 
 * -g -G  - debug options
 * 
 * for it's work:
 * export LD_LIBRARY_PATH=:/usr/local/cuda/lib
 * export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/usr/local/cuda/libnvvp/
 * 
 * cuda-gdb
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <memory.h>

#include <GL/freeglut.h>
#include <GL/gl.h>
#include <GL/glext.h>
#include <time.h>
#include <fstream>

#define cell_size			5
#define uchar				unsigned char
#define screen_width 		150
#define screen_height 		150

#define FIELD_WIDTH			10
#define FIELD_HEIGHT		10
#define NUMBER_OF_THREADS	10

//int width = screen_width*cell_size; //770; //1024;
//int height = screen_width*cell_size; //770; //768;

float * state_first;	// on PC
float * state_second;	// arrays

float * dev_first_state;	// on Card
float * dev_second_state;	// arrays

int * dev_width;
int * dev_height;

int width = FIELD_WIDTH;
int height = FIELD_HEIGHT;

uchar4 color1,color2;

void draw()
{
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, screen);
	glFlush();
}

void key(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27:
		printf("handled escape\nExit application\n");
		glutLeaveMainLoop();
		break;
	case ' ':
		transfer_cpu();
		draw_field();
		break;
	default:
		break;
	}
	draw();
}

void timer(int = 0)
{
	transfer_cpu();
	
	draw_field();
	draw();
	glutTimerFunc(200, timer, 0);
}

void start()
{
	gen();
	draw_field();
//	transfer_cpu();
	timer();
}

__global__ void kernel(float * first, float * second , int * width, int * height)
{
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	
	*(second + id) = 0;

	if(id <= (*width)*(*height))
	{
		int num = 0;
		
		// change to num += ...
		
		/*
		if(*(first + id + 1) == 1) num++;
		if(*(first + id - 1) == 1) num++;
		if(*(first + id + *height) == 1) num++;
		if(*(first + id - *height) == 1) num++;
		if(*(first + id + *height + 1) == 1) num++;
		if(*(first + id + *height - 1) == 1) num++;
		if(*(first + id - *height + 1) == 1) num++;
		if(*(first + id - *height - 1) == 1) num++;
		*/
		
		num += *(first + id + 1);
		num += *(first + id - 1);
		num += *(first + id + *height);
		num += *(first + id - *height);
		num += *(first + id + *height + 1);
		num += *(first + id + *height - 1);
		num += *(first + id - *height + 1);
		num += *(first + id - *height - 1);
		
		switch(num)
		{
			case 3 : *(second + id) = 1; break;
			case 2 : if(*(first + id) == 1) *(second + id) = 1; break;
			default : *(second + id) = 0; break;
		}
		
	}
}

void GetDataFromCudaDevice(int width, int height)
{
	hipMemcpy(state_first,dev_second_state,sizeof(float)*width*height,hipMemcpyDeviceToHost);
}

void CopyDataToCudaDevice(int width, int height)
{
	hipMemcpy(dev_first_state,state_first,sizeof(float)*width*height,hipMemcpyHostToDevice);
	hipMemset(dev_second_state,0,sizeof(float)*width*height);
	
	hipMemcpy(dev_width,&width,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_height,&height,sizeof(int),hipMemcpyHostToDevice);
}

void InitCudaArrays(int width, int height)
{
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}

	hipMalloc((void**)&dev_width,sizeof(int));
	hipMalloc((void**)&dev_height,sizeof(int));

	hipMalloc((void**)&dev_first_state,sizeof(float)*width*height);
	hipMalloc((void**)&dev_second_state,sizeof(float)*width*height);
}
// runs cuda device and returns result
void RunCudaDevice()
{
	hipError_t cudaStatus;

	int threads = NUMBER_OF_THREADS;
	int blocks = (width*height)/(NUMBER_OF_THREADS + 1);
	
	kernel <<<threads,blocks>>> (dev_first_state,dev_second_state,dev_width,dev_height);
//	kernel <<<10,10>>> (dev_first_state,dev_second_state,dev_width,dev_height);

	hipDeviceSynchronize();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return;
	}

	GetDataFromCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
}

void FreeCudaDevice(int width, int height)
{
	hipFree(dev_first_state);
	hipFree(dev_second_state);
	
	hipFree(dev_width);
	hipFree(dev_height);
}

void FillField()
{
	/*
	 *	
	 *	01010
	 *	00110
	 *	00100
	 *	00000
	 *	
	 */
	
//	state_first[9*width+9] = 1;

	state_first[2*width+5] = 1;
	state_first[2*width+6] = 1;
	state_first[3*width+6] = 1;
	state_first[3*width+7] = 1;
	state_first[1*width+7] = 1;

/*
	state_first[7*width+1] = 1;
	state_first[7*width+2] = 1;
	state_first[8*width+2] = 1;
	state_first[8*width+3] = 1;
	state_first[6*width+3] = 1;
*/
/*
	state_first[7*width+7] = 1;
	state_first[7*width+8] = 1;
	state_first[8*width+7] = 1;
	state_first[8*width+8] = 1;
*/
}
// allocate memory and initialize array with '0'
void InitArrays(int width, int height)
{
	state_first = (float *) malloc(sizeof(float)*width*height);
	state_second = (float *) malloc(sizeof(float)*width*height);
	
	memset(state_first,0,sizeof(float)*width*height);
	memset(state_second,0,sizeof(float)*width*height);
}

void ShowArray(int width, int height)
{
	puts("-----------------");
	for(int i=0;i<width;i++)
	{
		for(int j=0;j<height;j++)
		{
			if(state_first[i*width+j] != 0)printf("*");
			else printf(" ");
		//	printf("%1.0f",state_first[i*width+j]);
		}
		printf("\n");
	}
	puts("-----------------");
}

void CudaSwapArrays()
{
	float * t = dev_first_state;
	dev_first_state = dev_second_state;
	dev_second_state = t;
	
//	hipMemset(dev_second_state,0,sizeof(float)*width*height); //checking
}

void InitializwFreeGlut()
{
	// Initialize freeglut
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
	glutInitWindowSize(width, height);
	glutCreateWindow("Life");
	glutDisplayFunc(draw);
	glutKeyboardFunc(key);
	glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);
}

void init_colors()
{
	color1.x = 127;
	color1.y = 255;
	color1.z = 0;
	color1.w = 0;

	color2.x = 255;
	color2.y = 255;
	color2.z = 255;
	color2.w = 255;
}

int main()
{
	InitArrays(FIELD_WIDTH,FIELD_HEIGHT);
	
	FillField();
	
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	
	InitCudaArrays(FIELD_WIDTH,FIELD_HEIGHT);
	CopyDataToCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
	
	RunCudaDevice();
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	
	for(int i=0;i<30;i++)
	{
		CudaSwapArrays();
		RunCudaDevice();
		ShowArray(FIELD_WIDTH,FIELD_HEIGHT);
	}
	
	FreeCudaDevice(FIELD_WIDTH,FIELD_HEIGHT);
	
	ShowArray(FIELD_WIDTH,FIELD_HEIGHT);

//	char ch;
//	scanf("%c",&ch);
}
