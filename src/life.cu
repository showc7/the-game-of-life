#include "hip/hip_runtime.h"
/*
 *	THE GAME OF LIFE
 */


/*
 *
 * compiling:
 * nvcc -lglut -LGLEW life.cu -o life
 * 
 * for it's work:
 * export LD_LIBRARY_PATH=:/usr/local/cuda/lib
 * export LD_LIBRARY_PATH=$LD_LIBRARY_PATH:/usr/local/cuda/libnvvp/
 *
 * cuda-gdb
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <GL/freeglut.h>
#include <GL/gl.h>
#include <GL/glext.h>
#include <time.h>
#include <fstream>

#define cell_size 5
#define uchar unsigned char
#define screen_width 150
#define screen_height 150

int width = screen_width*cell_size; //770; //1024;
int height = screen_width*cell_size; //770; //768;

uchar4 * screen = NULL;
uchar field1[screen_width][screen_height];
uchar field2[screen_width][screen_height];

uchar4 color1,color2;

void draw(void)
{
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(width, height, GL_RGBA, GL_UNSIGNED_BYTE, screen);
	glFlush();
}

void draw_cube(int x, int y, int a, uchar4 color)
{
	for(int i=0;i<a;i++)
		for(int j=0;j<a;j++)
		{
			screen[(x+j)*width+y+i] = color;
		}
}

void draw_field()
{
	for(int i=0;i<screen_width;i++)
		for(int j=0;j<screen_height;j++)
		{
			if(field1[i][j] == 1)
				draw_cube(i*cell_size,j*cell_size,cell_size,color1);
			if(field1[i][j] == 0)
				draw_cube(i*cell_size,j*cell_size,cell_size,color2);
		}
}

void transfer_cpu()
{
	int num=0;
	
	for(int i=0;i<screen_width;i++)
		for(int j=0;j<screen_height;j++)
		{
			num = 0;
			
			if(i+1 < screen_width && field1[i+1][j] == 1) num++;
			if(i-1 >= 0 && field1[i-1][j] == 1) num++;
			if(j+1 < screen_height && field1[i][j+1] == 1) num++;
			if(j-1 >= 0 && field1[i][j-1] == 1) num++;
			if(i+1 < screen_width && j+1 < screen_height && field1[i+1][j+1] == 1) num++;
			if(i-1 > 0 && j+1 < screen_height && field1[i-1][j+1] == 1) num++;
			if(i+1 < screen_width && j-1 > 0 && field1[i+1][j-1] == 1) num++;
			if(i-1 > 0 && j-1 > 0 && field1[i-1][j-1] == 1) num++;
			
			switch(num)
			{
				case 3 : field2[i][j] = 1; break;
				case 2 : if(field1[i][j] == 1) field2[i][j] = 1; break;
				default : field2[i][j] = 0; break;
			}
		}
	
	for(int i=0;i<screen_width;i++)
		for(int j=0;j<screen_height;j++)
			field1[i][j] = field2[i][j];
}

void key(unsigned char key, int x, int y)
{
	switch (key)
	{
	case 27:
		printf("handled escape\nExit application\n");
		glutLeaveMainLoop();
		break;
	case ' ':
		transfer_cpu();
		draw_field();
		break;
	default:
		break;
	}
	draw();
}

void init_screen()
{
	screen = (uchar4 *) malloc(width * height * sizeof(uchar4));
	memset(screen, 0, width * height * sizeof(uchar4));
}

void field_to_zero(uchar * field)
{
	memset(field,0,height/cell_size*width/cell_size);
}

void gen()
{
	field1[0][0] = 1;
	field1[0][1] = 1;
	field1[0][2] = 1;
	
	field1[100][10] = 1;
	field1[100][11] = 1;
	field1[100][12] = 1;
	
	field1[7][10] = 1;
	field1[8][9] = 1;
	field1[8][8] = 1;
	field1[9][9] = 1;
	field1[9][10] = 1;
	
	
	field1[4][1] = 1;
	field1[2][2] = 1;
	field1[3][2] = 1;
	field1[3][3] = 1;
	field1[4][3] = 1;
	
	field1[40][10] = 1;
	field1[38][11] = 1;
	field1[39][11] = 1;
	field1[39][12] = 1;
	field1[40][12] = 1;
	
	field1[70][90] = 1;
	field1[70][91] = 1;
	field1[70][92] = 1;
	field1[69][89] = 1;
	field1[69][90] = 1;
	field1[69][91] = 1;
}

void timer(int = 0)
{
	transfer_cpu();
	draw_field();
	draw();
	glutTimerFunc(200, timer, 0);
}

void start()
{
	gen();
	draw_field();
//	transfer_cpu();
	timer();
}

void init_colors()
{
	color1.x = 127;
	color1.y = 255;
	color1.z = 0;
	color1.w = 0;
	
	color2.x = 255;
	color2.y = 255;
	color2.z = 255;
	color2.w = 255;
}

int main(int argc, char** argv)
{
	// Initialize freeglut
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_SINGLE | GLUT_RGBA);
	glutInitWindowSize(width, height);
	glutCreateWindow("Life");
	glutDisplayFunc(draw);
	glutKeyboardFunc(key);
	glutSetOption(GLUT_ACTION_ON_WINDOW_CLOSE, GLUT_ACTION_CONTINUE_EXECUTION);

	init_screen();
	
	// Initialization of colors
	init_colors();
	
	// Start of the program
	start();
	
	// Display Image
	glutMainLoop();

	// Free resources
	free(screen);
	screen = NULL;

	return 0;
}
